#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2021 Rohit Santhanam <rohit.santhanam@amd.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

template <typename>
void test_gpu_numext() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  bool* d_res_bfloat16 = (bool*)gpu_device.allocate(num_elem * sizeof(bool));
  bool* d_res_float = (bool*)gpu_device.allocate(num_elem * sizeof(bool));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_res_float(d_res_float, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_res_float.device(gpu_device) = gpu_float.unaryExpr(Eigen::internal::scalar_isnan_op<float>());
  // Test bfloat16 specific isnan op.
  gpu_res_bfloat16.device(gpu_device) =
      gpu_float.cast<Eigen::bfloat16>().unaryExpr(Eigen::internal::scalar_isnan_op<Eigen::bfloat16>());

  Tensor<bool, 1> bfloat16_prec(num_elem);
  Tensor<bool, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, num_elem * sizeof(bool));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem * sizeof(bool));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_EQUAL(full_prec(i), bfloat16_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

#ifdef EIGEN_HAS_GPU_BF16

template <typename>
void test_gpu_conversion() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::bfloat16* d_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  float* d_conv = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_bfloat16(d_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_conv(d_conv, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random();
  gpu_bfloat16.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>();
  gpu_conv.device(gpu_device) = gpu_bfloat16.cast<float>();

  Tensor<float, 1> initial(num_elem);
  Tensor<float, 1> final(num_elem);
  gpu_device.memcpyDeviceToHost(initial.data(), d_float, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(final.data(), d_conv, num_elem * sizeof(float));

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(static_cast<Eigen::bfloat16>(initial(i)), static_cast<Eigen::bfloat16>(final(i)));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_bfloat16);
  gpu_device.deallocate(d_conv);
}

template <typename>
void test_gpu_unary() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_bfloat16 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(d_res_float, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_float.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().cast<float>();
  gpu_res_float.device(gpu_device) = gpu_float.abs();
  gpu_res_bfloat16.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().abs().cast<float>();

  Tensor<float, 1> bfloat16_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem * sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(full_prec(i), bfloat16_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

template <typename>
void test_gpu_elementwise() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_bfloat16 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float1(d_float1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float2(d_float2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(d_res_float, num_elem);

  gpu_float1.device(gpu_device) = gpu_float1.random();
  gpu_float1.device(gpu_device) = gpu_float1.cast<Eigen::bfloat16>().cast<float>();
  gpu_float2.device(gpu_device) = gpu_float2.random();
  gpu_float2.device(gpu_device) = gpu_float2.cast<Eigen::bfloat16>().cast<float>();
  gpu_res_float.device(gpu_device) = (gpu_float1 + gpu_float2) * gpu_float1;
  gpu_res_bfloat16.device(gpu_device) =
      ((gpu_float1.cast<Eigen::bfloat16>() + gpu_float2.cast<Eigen::bfloat16>()) * gpu_float1.cast<Eigen::bfloat16>())
          .cast<float>();

  Tensor<float, 1> bfloat16_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem * sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(static_cast<Eigen::bfloat16>(full_prec(i)), static_cast<Eigen::bfloat16>(bfloat16_prec(i)));
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

template <typename>
void test_gpu_trancendental() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float3 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::bfloat16* d_res1_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res1_float = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res2_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res2_float = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res3_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res3_float = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float1(d_float1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float2(d_float2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float3(d_float3, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res1_bfloat16(d_res1_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res1_float(d_res1_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res2_bfloat16(d_res2_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res2_float(d_res2_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res3_bfloat16(d_res3_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res3_float(d_res3_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res4_bfloat16(d_res3_bfloat16, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res4_float(d_res3_float, num_elem);

  gpu_float1.device(gpu_device) = gpu_float1.random() - gpu_float1.constant(0.5f);
  gpu_float1.device(gpu_device) = gpu_float1.cast<Eigen::bfloat16>().cast<float>();
  gpu_float2.device(gpu_device) = gpu_float2.random() + gpu_float1.constant(0.5f);
  gpu_float2.device(gpu_device) = gpu_float2.cast<Eigen::bfloat16>().cast<float>();
  gpu_float3.device(gpu_device) = gpu_float3.random();
  gpu_float3.device(gpu_device) = gpu_float3.cast<Eigen::bfloat16>().cast<float>();
  gpu_res1_float.device(gpu_device) = gpu_float1.exp().cast<Eigen::bfloat16>();
  gpu_res2_float.device(gpu_device) = gpu_float2.log().cast<Eigen::bfloat16>();
  gpu_res3_float.device(gpu_device) = gpu_float3.log1p().cast<Eigen::bfloat16>();
  gpu_res4_float.device(gpu_device) = gpu_float3.expm1().cast<Eigen::bfloat16>();

  gpu_res1_bfloat16.device(gpu_device) = gpu_float1.cast<Eigen::bfloat16>();
  gpu_res1_bfloat16.device(gpu_device) = gpu_res1_bfloat16.exp();

  gpu_res2_bfloat16.device(gpu_device) = gpu_float2.cast<Eigen::bfloat16>();
  gpu_res2_bfloat16.device(gpu_device) = gpu_res2_bfloat16.log();

  gpu_res3_bfloat16.device(gpu_device) = gpu_float3.cast<Eigen::bfloat16>();
  gpu_res3_bfloat16.device(gpu_device) = gpu_res3_bfloat16.log1p();

  gpu_res3_bfloat16.device(gpu_device) = gpu_float3.cast<Eigen::bfloat16>();
  gpu_res3_bfloat16.device(gpu_device) = gpu_res3_bfloat16.expm1();

  Tensor<float, 1> input1(num_elem);
  Tensor<Eigen::bfloat16, 1> bfloat16_prec1(num_elem);
  Tensor<Eigen::bfloat16, 1> full_prec1(num_elem);
  Tensor<float, 1> input2(num_elem);
  Tensor<Eigen::bfloat16, 1> bfloat16_prec2(num_elem);
  Tensor<Eigen::bfloat16, 1> full_prec2(num_elem);
  Tensor<float, 1> input3(num_elem);
  Tensor<Eigen::bfloat16, 1> bfloat16_prec3(num_elem);
  Tensor<Eigen::bfloat16, 1> full_prec3(num_elem);
  gpu_device.memcpyDeviceToHost(input1.data(), d_float1, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(input2.data(), d_float2, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(input3.data(), d_float3, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(bfloat16_prec1.data(), d_res1_bfloat16, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec1.data(), d_res1_float, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(bfloat16_prec2.data(), d_res2_bfloat16, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec2.data(), d_res2_float, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(bfloat16_prec3.data(), d_res3_bfloat16, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec3.data(), d_res3_float, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(full_prec1(i), bfloat16_prec1(i));
  }
  for (int i = 0; i < num_elem; ++i) {
    if (std::abs(input2(i) - 1.f) < 0.05f)  // log lacks accuracy nearby 1
      VERIFY_IS_APPROX(full_prec2(i) + Eigen::bfloat16(0.1f), bfloat16_prec2(i) + Eigen::bfloat16(0.1f));
    else
      VERIFY_IS_APPROX(full_prec2(i), bfloat16_prec2(i));
  }
  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(full_prec3(i), bfloat16_prec3(i));
  }
  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_float3);
  gpu_device.deallocate(d_res1_bfloat16);
  gpu_device.deallocate(d_res1_float);
  gpu_device.deallocate(d_res2_bfloat16);
  gpu_device.deallocate(d_res2_float);
  gpu_device.deallocate(d_res3_float);
  gpu_device.deallocate(d_res3_bfloat16);
}

template <typename>
void test_gpu_contractions() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int rows = 23;
  int cols = 23;
  int num_elem = rows * cols;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::bfloat16* d_res_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res_float = (Eigen::bfloat16*)gpu_device.allocate(num_elem * sizeof(Eigen::bfloat16));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(d_float1, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(d_float2, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 2>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 2>, Eigen::Aligned> gpu_res_float(d_res_float, rows, cols);

  gpu_float1.device(gpu_device) = gpu_float1.random() - gpu_float1.constant(0.5f);
  gpu_float2.device(gpu_device) = gpu_float2.random() - gpu_float2.constant(0.5f);

  typedef Tensor<float, 2>::DimensionPair DimPair;
  Eigen::array<DimPair, 1> dims(DimPair(1, 0));
  gpu_res_float.device(gpu_device) = gpu_float1.contract(gpu_float2, dims).cast<Eigen::bfloat16>();
  gpu_res_bfloat16.device(gpu_device) =
      gpu_float1.cast<Eigen::bfloat16>().contract(gpu_float2.cast<Eigen::bfloat16>(), dims);

  Tensor<Eigen::bfloat16, 2> bfloat16_prec(rows, cols);
  Tensor<Eigen::bfloat16, 2> full_prec(rows, cols);
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem * sizeof(Eigen::bfloat16));
  gpu_device.synchronize();

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      if (numext::abs(full_prec(i, j) - bfloat16_prec(i, j)) > Eigen::bfloat16(1e-2f)) {
        VERIFY_IS_APPROX(full_prec(i, j), bfloat16_prec(i, j));
      }
    }
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

template <typename>
void test_gpu_reductions(int size1, int size2, int redux) {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = size1 * size2;
  int result_size = (redux == 1 ? size1 : size2);

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::bfloat16* d_res_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(result_size * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res_float = (Eigen::bfloat16*)gpu_device.allocate(result_size * sizeof(Eigen::bfloat16));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float(d_float, size1, size2);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16, result_size);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 1>, Eigen::Aligned> gpu_res_float(d_res_float, result_size);

  gpu_float.device(gpu_device) = gpu_float.random() * 2.0f;

  Eigen::array<int, 1> redux_dim = {redux};
  gpu_res_float.device(gpu_device) = gpu_float.sum(redux_dim).cast<Eigen::bfloat16>();
  gpu_res_bfloat16.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().sum(redux_dim);

  Tensor<Eigen::bfloat16, 1> bfloat16_prec(result_size);
  Tensor<Eigen::bfloat16, 1> full_prec(result_size);
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, result_size * sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, result_size * sizeof(Eigen::bfloat16));
  gpu_device.synchronize();

  for (int i = 0; i < result_size; ++i) {
    VERIFY_IS_APPROX(full_prec(i), bfloat16_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

template <typename>
void test_gpu_reductions() {
  test_gpu_reductions<void>(13, 13, 0);
  test_gpu_reductions<void>(13, 13, 1);

  test_gpu_reductions<void>(35, 36, 0);
  test_gpu_reductions<void>(35, 36, 1);

  test_gpu_reductions<void>(36, 35, 0);
  test_gpu_reductions<void>(36, 35, 1);
}

template <typename>
void test_gpu_full_reductions() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int size = 13;
  int num_elem = size * size;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::bfloat16* d_res_bfloat16 = (Eigen::bfloat16*)gpu_device.allocate(1 * sizeof(Eigen::bfloat16));
  Eigen::bfloat16* d_res_float = (Eigen::bfloat16*)gpu_device.allocate(1 * sizeof(Eigen::bfloat16));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float(d_float, size, size);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 0>, Eigen::Aligned> gpu_res_bfloat16(d_res_bfloat16);
  Eigen::TensorMap<Eigen::Tensor<Eigen::bfloat16, 0>, Eigen::Aligned> gpu_res_float(d_res_float);

  gpu_float.device(gpu_device) = gpu_float.random();

  gpu_res_float.device(gpu_device) = gpu_float.sum().cast<Eigen::bfloat16>();
  gpu_res_bfloat16.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().sum();

  Tensor<Eigen::bfloat16, 0> bfloat16_prec;
  Tensor<Eigen::bfloat16, 0> full_prec;
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, sizeof(Eigen::bfloat16));
  gpu_device.synchronize();

  VERIFY_IS_APPROX(full_prec(), bfloat16_prec());

  gpu_res_float.device(gpu_device) = gpu_float.maximum().cast<Eigen::bfloat16>();
  gpu_res_bfloat16.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().maximum();
  gpu_device.memcpyDeviceToHost(bfloat16_prec.data(), d_res_bfloat16, sizeof(Eigen::bfloat16));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, sizeof(Eigen::bfloat16));
  gpu_device.synchronize();

  VERIFY_IS_APPROX(full_prec(), bfloat16_prec());

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_bfloat16);
  gpu_device.deallocate(d_res_float);
}

template <typename>
void test_gpu_forced_evals() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_bfloat16_1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_bfloat16_2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_bfloat16_1(d_res_bfloat16_1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Unaligned> gpu_res_bfloat16_2(d_res_bfloat16_2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(d_res_float, num_elem);

  Eigen::array<int, 1> no_bcast;
  no_bcast[0] = 1;

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_float.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().cast<float>();
  gpu_res_float.device(gpu_device) = gpu_float.abs();
  gpu_res_bfloat16_1.device(gpu_device) = gpu_float.cast<Eigen::bfloat16>().abs().eval().cast<float>();
  gpu_res_bfloat16_2.device(gpu_device) =
      gpu_float.cast<Eigen::bfloat16>().abs().broadcast(no_bcast).eval().cast<float>();

  Tensor<float, 1> bfloat16_prec1(num_elem);
  Tensor<float, 1> bfloat16_prec2(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(bfloat16_prec1.data(), d_res_bfloat16_1, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(bfloat16_prec2.data(), d_res_bfloat16_2, num_elem * sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem * sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(full_prec(i), bfloat16_prec1(i));
    VERIFY_IS_APPROX(full_prec(i), bfloat16_prec2(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_bfloat16_1);
  gpu_device.deallocate(d_res_bfloat16_2);
  gpu_device.deallocate(d_res_float);
}

#endif

EIGEN_DECLARE_TEST(cxx11_tensor_of_bfloat16_gpu) {
  CALL_SUBTEST_1(test_gpu_numext<void>());

// The reduction unit tests have been excluded until a working
// implementation to expand the accumulator data type to float32
// is available.
// TODO: add reduction unit tests
#ifdef EIGEN_HAS_GPU_BF16
  CALL_SUBTEST_2(test_gpu_conversion<void>());
  CALL_SUBTEST_3(test_gpu_unary<void>());
  CALL_SUBTEST_4(test_gpu_elementwise<void>());
  CALL_SUBTEST_5(test_gpu_trancendental<void>());
  CALL_SUBTEST_6(test_gpu_contractions<void>());
  CALL_SUBTEST_7(test_gpu_reductions<void>());
  CALL_SUBTEST_8(test_gpu_full_reductions<void>());
  CALL_SUBTEST_9(test_gpu_forced_evals<void>());
#else
  std::cout << "bfloat16 floats are not supported by this version of gpu: skipping the test" << std::endl;
#endif
}
